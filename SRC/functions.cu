#include "hip/hip_runtime.h"

extern "C" {
    #include "functions.h"
    #include "mt.h"
}

__constant__ int d_nmol;

/***   Timer   ***/
void initTimer() {
	t_i = time(NULL);
	gettimeofday(&last_t, NULL);
}

void finTimer() {
	delta_t = time(NULL) - t_i;
    //printf("%lf", (double)delta_t);
}

void timeval_subtract (struct timeval *result, struct timeval *x, struct timeval *y) {
	int nsec;

	/* Perform the carry for the later subtraction by updating y. */
	if (x->tv_usec < y->tv_usec) {
		nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
		y->tv_usec -= 1000000 * nsec;
		y->tv_sec += nsec;
	}
	if (x->tv_usec - y->tv_usec > 1000000) {
		nsec = (x->tv_usec - y->tv_usec) / 1000000;
		y->tv_usec += 1000000 * nsec;
		y->tv_sec -= nsec;
	}

	/* Compute the time remaining to wait. tv_usec is certainly positive. */
	result->tv_sec = x->tv_sec - y->tv_sec;
	result->tv_usec = x->tv_usec - y->tv_usec;
}

/***   Files   ***/
int getParameter(const char *field_name, const char *token, const char *format, void *variable, FILE *fi) {
	char line[SEQ_MAX_LENGTH+12];
	char *p_token, *p_aux;
	int flag_done = 0;
	int len, i;

	do {
		if(fgets(line, SEQ_MAX_LENGTH+12, fi) == 0)
			return FAIL;

		//line[SEQ_MAX_LENGTH+10] = '\0';
		len = strlen(line);
		if(line[len-1] == '\n')
			line[len-1] = '\0';

		switch(flag_done) {
			case 0:
				if(strstr(line, "/*") != 0) {
					flag_done = 1;
					if(strstr(line, "*/") != 0)
						flag_done = 2;
				}
				break;
			case 1:
				if(strstr(line, "*/") != 0)
					flag_done = 2;
				break;
			case 2:
				if(strstr(line, "/*") != 0) {
					flag_done = 1;
					if(strstr(line, "*/") != 0)
						flag_done = 2;
				} else {
					flag_done = 0;
				}
				break;
		}
	} while(flag_done != 0);

	if(strstr(line, field_name) != 0) {
		p_token = strtok(line, token);

		while(p_token != NULL) {
			p_aux = p_token;
			p_token = strtok(NULL, token);
		}

		if(p_aux[0] == ' ') {

			for(i = 0; i < len - 1; i++) {
				p_aux[i] = p_aux[i + 1];
			}
		}
		//puts(p_aux);
		sscanf(p_aux, format, variable);
		return SUCCESS;

	} else {
		return FAIL;
	}
}

int loadFile(char **argv) {
	FILE *fi;
    	fi = fopen(argv[1], "r");

	if(fi != 0) {
		getParameter("sequence", "=", "%s", sequence, fi);
		getParameter("ProtLen", "=", "%d", &prot_len, fi);		
		getParameter("LV", "=", "%lf", &LV, fi);
		getParameter("stepLimit", "=", "%d", &step_limit, fi);
		getParameter("temperature", "=", "%lf", &temperature, fi);
		getParameter("savepathways", "=", "%c", &save_pathways, fi);
		getParameter("pathwaysstep", "=", "%d", &pathways_step, fi);

		fclose(fi);

		mass = 1.0; 
		n_mol = prot_len;
		bond_len = 1.0;
		n_c = (prot_len-1);
		dt = 0.0001;
		r_cut = 2.38;
		c_T = 0.01;
		display_interval = 100;
		step2resc_vels = 1;
		temperature_steps = 0.10;
		report_file = 'y';
		step2report = 'n';
		print_summary = 160;
		print_summary_interval = 'y';
		print_summary2file = 'y';
		shake_cons_prec = 1.0e-06;
		shake_max_cycle = 1; 
		shake_step2shake = 10;

				

        if(n_mol < 1025)
		{
			if(n_mol > 512) {
                blockSize = 1024;
            } else {
                if(n_mol > 256) {
                    blockSize = 512;
                } else {
                    if(n_mol > 128) {
                        blockSize = 256;
                    } else {
                        if(n_mol > 64) {
                            blockSize = 128;
                        } else {
                            if(n_mol > 32) {
                                blockSize = 64;
                            } else {
                                blockSize = 32;
                            }
                        }
                    }
                }
            }
        } 
		else {
            return FAIL;
        }

        if(print_summary != 'y' && print_summary != 'Y') print_summary_interval = step_limit - 1;
        if(report_file != 'y' && report_file != 'Y') step2report = step_limit - 1;
        if(save_pathways != 'y' && save_pathways != 'Y') pathways_step = step_limit - 1;
        en_update = min(print_summary_interval, step2report);
        en_update = min(en_update, pathways_step);

		return SUCCESS;
	} else {
		return FAIL;
	}
}

void putParameters() {
	printf("Sequence >>                         %s\n", sequence);
	printf("Mass >>                             %lf\n", mass);
	printf("Number of Particles >>              %d\n", n_mol);
	printf("Bond Length >>                      %d\n", bond_len);
	printf("Protein Length >>                   %d\n", prot_len);
	printf("Number of Constraints >>            %d\n", n_c);
	printf("Dimension of the Box >>             %lf\n", LV);
	printf("Time Step >>                        %lf\n", dt);
	printf("Cutoff Radius >>                    %lf\n", r_cut);
	printf("Skin >>                             %lf\n", skin);
	printf("Steps to Update Neighbour List >>   %d\n", step2neighbour);
	printf("cT >>                               %lf\n", c_T);
	printf("Display Interval >>                 %d\n", display_interval);
	printf("Step Limit >>                       %d\n", step_limit);
	printf("Steps to Rescale Velocities >>      %d\n", step2resc_vels);
	printf("Temperature >>                      %lf\n", temperature);
	printf("Temperature Steps >>                %lf\n", temperature_steps);
	printf("Report File >>                      %c\n", report_file);
	printf("Steps to Report >>                  %d\n", step2report);
	printf("Print Summary >>                    %c\n", print_summary);
	printf("Print Summary Interval >>           %d\n", print_summary_interval);
	printf("Print Summary to File >>            %c\n", print_summary2file);
	printf("Shake-consPrec >>                   %lf\n", shake_cons_prec);
	printf("Shake-maxCycle >>                   %d\n", shake_max_cycle);
	printf("Steps to Shake >>                   %d\n", shake_step2shake);
	printf("Save Pathways >>                    %c\n", save_pathways);
	printf("Pathways Step >>                    %d\n", pathways_step);
}

/***   Utilities   ***/
double randdouble(double max) {
	double ret;


	ret = fabs(((randomMT() + (RAND_MAX_MT/2)) / ((double)(RAND_MAX_MT)) * max));

	return ret;
}

int isUnique(Particle *p, int last) {
	int i;
	for(i = 0; i < last; i++) {
		if((p[i].v_r.x == p[last].v_r.x) && (p[i].v_r.y == p[last].v_r.y) && (p[i].v_r.z == p[last].v_r.z))
			return 0;
	}
	return 1;
}

void verifyBoundary1(VectorR *v) {
	if(v->x < 0) {
		v->x += LV;
	} else {
		if(v->x >= LV)
			v->x -= LV;
	}

	if(v->y < 0) {
		v->y += LV;
	} else {
		if(v->y >= LV)
			v->y -= LV;
	}

	if(v->z < 0) {
		v->z += LV;
	} else {
		if(v->z >= LV)
			v->z -= LV;
	}
}

void verifyBoundary2(VectorR *v) {
	if(v->x >= 0.5 * LV) {
		v->x -= LV;
	} else {
		if(v->x < -0.5 * LV)
			v->x += LV;
	}

	if(v->y >= 0.5 * LV) {
		v->y -= LV;
	} else {
		if(v->y < -0.5 * LV)
			v->y += LV;
	}

	if(v->z >= 0.5 * LV) {
		v->z -= LV;
	} else {
		if(v->z < -0.5 * LV)
			v->z += LV;
	}
}

__device__ void cudaVerifyBoundary1(VectorR *v, double LV) {
	if(v->x < 0) {
		v->x += LV;
	} else {
		if(v->x >= LV)
			v->x -= LV;
	}

	if(v->y < 0) {
		v->y += LV;
	} else {
		if(v->y >= LV)
			v->y -= LV;
	}

	if(v->z < 0) {
		v->z += LV;
	} else {
		if(v->z >= LV)
			v->z -= LV;
	}
}

__device__ void cudaVerifyBoundary2(VectorR *v, double LV) {
    if(v->x >= 0.5 * LV) {
		v->x -= LV;
	} else {
		if(v->x < -0.5 * LV)
			v->x += LV;
	}

	if(v->y >= 0.5 * LV) {
		v->y -= LV;
	} else {
		if(v->y < -0.5 * LV)
			v->y += LV;
	}

	if(v->z >= 0.5 * LV) {
		v->z -= LV;
	} else {
		if(v->z < -0.5 * LV)
			v->z += LV;
	}
}

/***   Initialize   ***/
void allocArrays() {
	int i;

	particles = (Particle *) malloc(n_mol * sizeof(Particle));

	best_structure = (Particle *) malloc(n_mol * sizeof(Particle));

	constraint = (Constraint *) malloc(n_c * sizeof(Constraint));

	mMat = (int **) malloc(n_mol * sizeof(int *));
	mMat[0] = (int *) malloc(n_mol * n_c * sizeof(int));
	for(i = 1; i < n_mol; i++)
		mMat[i] = mMat[0] + i * n_c;

	
}

void allocDevice() {
	hipMemcpyToSymbol(HIP_SYMBOL(d_nmol), &n_mol, sizeof(int));
    hipMalloc((void **)&d_particles, blockSize * sizeof(Particle));
    hipMalloc((void **)&d_uB, sizeof(double));
    hipMalloc((void **)&d_uT, sizeof(double));
    hipMalloc((void **)&d_uLJ, sizeof(double));
    hipMalloc((void **)&d_uLJVector, blockSize * sizeof(double));
    hipMalloc((void **)&d_sequence, n_mol * sizeof(char));
    hipMalloc((void **)&d_neighbour_list, n_mol * n_mol * sizeof(int));
    hipMalloc((void**)&d_constraint, n_c * sizeof(Constraint));
}

void setVelMag() {
	vel_mag = sqrt(N_DIM * (1. - 1./prot_len) * temperature);
}

void initCoords() {
	int i, j, conf_OK;
	double theta;
	double phi;
    double dist;

	particles[0].v_r.x = LV/2;
	particles[0].v_r.y = LV/2;
	particles[0].v_r.z = LV/2;
	//printf("(%lf, %lf, %lf)\n", particles[0].v_r.x, particles[0].v_r.y, particles[0].v_r.z);
	for(i = 1;i < n_mol; i++) {
		do {
			theta = randdouble(180.0) * M_PI / 180.0;
			phi = randdouble(360.0) * M_PI / 180.0;


			particles[i].v_r.x = particles[i-1].v_r.x + sin(theta) * cos(phi);
			particles[i].v_r.y = particles[i-1].v_r.y + sin(theta) * sin(phi);
			particles[i].v_r.z = particles[i-1].v_r.z + cos(theta);
            //printf("(%lf, %lf, %lf) theta %lf\n", particles[i].v_r.x, particles[i].v_r.y, particles[i].v_r.z, theta);

			verifyBoundary1(&particles[i].v_r);

            conf_OK = 1; //OK
			for (j = 0; j < i; j++)
			{
				dist = sqrt(sqr(particles[i].v_r.x - particles[j].v_r.x) + sqr(particles[i].v_r.y - particles[j].v_r.y) + sqr(particles[i].v_r.z - particles[j].v_r.z));
				if (dist < 1)
					conf_OK = 0; 
			}
            //printf("\t(%lf, %lf, %lf)\n", particles[i].v_r.x, particles[i].v_r.y, particles[i].v_r.z);
		} while((isUnique(particles, i) != 1) || (conf_OK != 1));
	}
}

void initVels() {
	int i;
	double theta;
	double phi;
	VectorR sum_v;

	sum_v.x = 0;
	sum_v.y = 0;
	sum_v.z = 0;

	/***   Generating Vectors   ***/
	setVelMag();
	for(i = 0; i < n_mol; i++) {
		theta = randdouble(180.0) * M_PI / 180.0;
		phi = randdouble(360.0) * M_PI / 360.0;

		particles[i].v_v.x = sin(theta) * cos(phi) * vel_mag;
		particles[i].v_v.y = sin(theta) * sin(phi) * vel_mag;
		particles[i].v_v.z = cos(theta) * vel_mag;
        //printf("(%lf, %lf, %lf)\n", particles[i].v_v.x/vel_mag, particles[i].v_v.y/vel_mag, particles[i].v_v.z/vel_mag);

		sum_v.x += particles[i].v_v.x;
		sum_v.y += particles[i].v_v.y;
		sum_v.z += particles[i].v_v.z;

	}

	/***   Center of Mass at Rest   ***/
	for(i = 0; i < n_mol; i++) {
		particles[i].v_v.x = particles[i].v_v.x - (sum_v.x / prot_len);
		particles[i].v_v.y = particles[i].v_v.y - (sum_v.y / prot_len);
		particles[i].v_v.z = particles[i].v_v.z - (sum_v.z / prot_len);

	}
}

void initAccs() {
	int i;

	for(i = 0; i < n_mol; i++) {
		particles[i].v_a.x = 0.;
		particles[i].v_a.y = 0.;
		particles[i].v_a.z = 0.;
    }
}

void buildConstMatrix() {
	int i, j;

	for(i = 0; i < n_mol; i++)
		for(j = 0; j < n_c; j++)
			mMat[i][j] = 0;

	for(i = 0; i < n_mol; i++) {
		if(i - 1 > 0)
			mMat[i - 1][i] = 2;
		if(i < n_c)
			mMat[i][i] = -2;
	}

	for(i = 0; i < n_c; i++) {
		/***   Distance sqr taken as 1   ***/
		constraint[i].ik = i;
		constraint[i].jk = i + 1;

        //printf("%d %d\n", constraint[i].ik, constraint[i].jk);
	}
}

void initMD() {
	allocArrays();
    allocDevice();
	initCoords();
	initVels();
	initAccs();
	buildConstMatrix();

    hipMemcpy(d_sequence, sequence, n_mol * sizeof(char), hipMemcpyHostToDevice);

}

/***   MD   ***/
__global__ void cudaUpdatePos(Particle *particles, double dt ,double LV) {
	//int threadId = threadIdx.x;

	for(int threadId = threadIdx.x; threadId < d_nmol; threadId += blockDim.x){
		particles[threadId].v_r.x += particles[threadId].v_v.x * dt + 0.5 * particles[threadId].v_a.x * sqr(dt);
		particles[threadId].v_r.y += particles[threadId].v_v.y * dt + 0.5 * particles[threadId].v_a.y * sqr(dt);
		particles[threadId].v_r.z += particles[threadId].v_v.z * dt + 0.5 * particles[threadId].v_a.z * sqr(dt);

        //cudaVerifyBoundary1(&particles[i].v_r);
        if(particles[threadId].v_r.x < 0) {
    		particles[threadId].v_r.x += LV;
    	} else {
    		if(particles[threadId].v_r.x >= LV)
    			particles[threadId].v_r.x -= LV;
    	}

    	if(particles[threadId].v_r.y < 0) {
    		particles[threadId].v_r.y += LV;
    	} else {
    		if(particles[threadId].v_r.y >= LV)
    			particles[threadId].v_r.y -= LV;
    	}

    	if(particles[threadId].v_r.z < 0) {
    		particles[threadId].v_r.z += LV;
    	} else {
    		if(particles[threadId].v_r.z >= LV)
    			particles[threadId].v_r.z -= LV;
    	}

        // First part of the velocity verlet
		particles[threadId].v_v.x += 0.5 * dt * particles[threadId].v_a.x;
		particles[threadId].v_v.y += 0.5 * dt * particles[threadId].v_a.y;
		particles[threadId].v_v.z += 0.5 * dt * particles[threadId].v_a.z;

        // Initialize the step acceleration
		particles[threadId].v_a.x = 0.;
		particles[threadId].v_a.y = 0.;
		particles[threadId].v_a.z = 0.;

	}
}


__global__ void cudaUpdateNeighbourList(Particle *particles, int *neighbour_list, double r_cut, double skin) {

	for(int i = blockIdx.x; i < d_nmol - 2; i += gridDim.x) {
		for(int j = threadIdx.x; j >= i + 2 && j < d_nmol; j += blockDim.x) {
			double r = sqrt(sqr(particles[j].v_r.x - particles[i].v_r.x) + sqr(particles[j].v_r.y - particles[i].v_r.y) + sqr(particles[j].v_r.z - particles[i].v_r.z));

			if(r <= (r_cut + skin))
				neighbour_list[i * d_nmol + j] = 1;
			else
            	neighbour_list[i * d_nmol + j] = 0;
		}
	}
}

__global__ void cudaBondEnergy(Particle *particles, double *d_uB, double LV) {
    // Restrict to one block. Limit: 1024 threads/amino acids (compute capability >= 2.x)
    extern __shared__ double d_uBond[];
    int blockSize = blockDim.x;
    int threadId = threadIdx.x;
    VectorR dr1, dr2, a1, a2;
    double c11, c12, c22, cd, f;
    //double c

    if(threadId < (d_nmol - 2)) {
        dr1.x = particles[threadId + 1].v_r.x - particles[threadId].v_r.x;
        dr1.y = particles[threadId + 1].v_r.y - particles[threadId].v_r.y;
        dr1.z = particles[threadId + 1].v_r.z - particles[threadId].v_r.z;
        dr2.x = particles[threadId + 2].v_r.x - particles[threadId + 1].v_r.x;
        dr2.y = particles[threadId + 2].v_r.y - particles[threadId + 1].v_r.y;
        dr2.z = particles[threadId + 2].v_r.z - particles[threadId + 1].v_r.z;

        cudaVerifyBoundary2(&dr1, LV);
        cudaVerifyBoundary2(&dr2, LV);

        c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
        c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
        c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;

        cd = sqrt (c11 * c22);


        d_uBond[threadId] = c12;

        f = -1.0;
        a1.x = (f / cd) * ((c12 / c11) * dr1.x  - dr2.x);
        a1.y = (f / cd) * ((c12 / c11) * dr1.y  - dr2.y);
        a1.z = (f / cd) * ((c12 / c11) * dr1.z  - dr2.z);
        a2.x = (f / cd) * (dr1.x - (c12 / c22) * dr2.x);
        a2.y = (f / cd) * (dr1.y - (c12 / c22) * dr2.y);
        a2.z = (f / cd) * (dr1.z - (c12 / c22) * dr2.z);

        particles[threadId + 2].v_a.x = 0;
        particles[threadId + 2].v_a.y = 0;
        particles[threadId + 2].v_a.z = 0;
        __syncthreads();
        particles[threadId].v_a.x = a1.x;
        particles[threadId].v_a.y = a1.y;
        particles[threadId].v_a.z = a1.z;
        __syncthreads();
        particles[threadId + 1].v_a.x -= a1.x;
        particles[threadId + 1].v_a.y -= a1.y;
        particles[threadId + 1].v_a.z -= a1.z;
        particles[threadId + 1].v_a.x -= a2.x;
        particles[threadId + 1].v_a.y -= a2.y;
        particles[threadId + 1].v_a.z -= a2.z;
        __syncthreads();
        particles[threadId + 2].v_a.x += a2.x;
        particles[threadId + 2].v_a.y += a2.y;
        particles[threadId + 2].v_a.z += a2.z;

    } else {
        d_uBond[threadId] = 0;
    }

    if (blockSize >= 512) {
        if (threadId < 256)
            d_uBond[threadId] += d_uBond[threadId + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (threadId < 128)
            d_uBond[threadId] += d_uBond[threadId + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (threadId < 64)
            d_uBond[threadId] += d_uBond[threadId + 64];
        __syncthreads();
    }

    if (threadId < 32) {
        if (blockSize >= 64) d_uBond[threadId] += d_uBond[threadId + 32];
        if (blockSize >= 32) d_uBond[threadId] += d_uBond[threadId + 16];
        if (blockSize >= 16) d_uBond[threadId] += d_uBond[threadId + 8];
        if (blockSize >= 8) d_uBond[threadId] += d_uBond[threadId + 4];
        if (blockSize >= 4) d_uBond[threadId] += d_uBond[threadId + 2];
        if (blockSize >= 2) d_uBond[threadId] += d_uBond[threadId + 1];

        if(threadId == 0) {
            *d_uB = d_uBond[0];
        }
    }

}

__global__ void cudaTorsionEnergy(Particle *particles, double *d_uT, double LV) {
    extern __shared__ double d_uTorsion[];
    int blockSize = blockDim.x;
    int threadId = threadIdx.x;
    VectorR dr1, dr2, dr3, a1, a2;
    double c11, c12, c13, c22, c23, c33, pi, qia, qib, cr1, cr2, t1, t2, t3, t4, t5, t6, f;

    if(threadId < (d_nmol - 3)) {
        dr1.x = particles[threadId + 1].v_r.x - particles[threadId].v_r.x;
        dr1.y = particles[threadId + 1].v_r.y - particles[threadId].v_r.y;
        dr1.z = particles[threadId + 1].v_r.z - particles[threadId].v_r.z;
        dr2.x = particles[threadId + 2].v_r.x - particles[threadId + 1].v_r.x;
        dr2.y = particles[threadId + 2].v_r.y - particles[threadId + 1].v_r.y;
        dr2.z = particles[threadId + 2].v_r.z - particles[threadId + 1].v_r.z;
        dr3.x = particles[threadId + 3].v_r.x - particles[threadId + 2].v_r.x;
        dr3.y = particles[threadId + 3].v_r.y - particles[threadId + 2].v_r.y;
        dr3.z = particles[threadId + 3].v_r.z - particles[threadId + 2].v_r.z;

        cudaVerifyBoundary2(&dr1, LV);
        cudaVerifyBoundary2(&dr2, LV);
        cudaVerifyBoundary2(&dr3, LV);

        c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
        c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
        c13 = dr1.x * dr3.x + dr1.y * dr3.y + dr1.z * dr3.z;
        c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;
        c23 = dr2.x * dr3.x + dr2.y * dr3.y + dr2.z * dr3.z;
        c33 = dr3.x * dr3.x + dr3.y * dr3.y + dr3.z * dr3.z;

        pi = c13 * c22 - c12 * c23;
        qia = c11 * c22 - c12 * c12;
        qib = c22 * c33 - c23 * c23;
        cr1 = c12 / c22;
        cr2 = c23 / c22;

        t1 = pi;
        t2 = c11 * c23 - c12 * c13;
        t3 = - qia;
        t4 = qib;
        t5 = c13 * c23 - c12 * c33;
        t6 = - t1;

        d_uTorsion[threadId] = (- 0.5) * c13;

        f = 0.5;

        a1.x = f * c22 * (t1 * dr1.x + t2 * dr2.x + t3 * dr3.x) / (sqrt(qia * qib) * qia);
        a1.y = f * c22 * (t1 * dr1.y + t2 * dr2.y + t3 * dr3.y) / (sqrt(qia * qib) * qia);
        a1.z = f * c22 * (t1 * dr1.z + t2 * dr2.z + t3 * dr3.z) / (sqrt(qia * qib) * qia);
        a2.x = f * c22 * (t4 * dr1.x + t5 * dr2.x + t6 * dr3.x) / (sqrt(qia * qib) * qib);
        a2.y = f * c22 * (t4 * dr1.y + t5 * dr2.y + t6 * dr3.y) / (sqrt(qia * qib) * qib);
        a2.z = f * c22 * (t4 * dr1.z + t5 * dr2.z + t6 * dr3.z) / (sqrt(qia * qib) * qib);

        particles[threadId].v_a.x += a1.x;
        particles[threadId].v_a.y += a1.y;
        particles[threadId].v_a.z += a1.z;
        __syncthreads();
        particles[threadId + 1].v_a.x += (- (1. + cr1) * a1.x) + cr2 * a2.x;
        particles[threadId + 1].v_a.y += (- (1. + cr1) * a1.y) + cr2 * a2.y;
        particles[threadId + 1].v_a.z += (- (1. + cr1) * a1.z) + cr2 * a2.z;
        __syncthreads();
        particles[threadId + 2].v_a.x += cr1 * a1.x + (- (1. + cr2)) * a2.x;
        particles[threadId + 2].v_a.y += cr1 * a1.y + (- (1. + cr2)) * a2.y;
        particles[threadId + 2].v_a.z += cr1 * a1.z + (- (1. + cr2)) * a2.z;
        __syncthreads();
        particles[threadId + 3].v_a.x += a2.x;
        particles[threadId + 3].v_a.y += a2.y;
        particles[threadId + 3].v_a.z += a2.z;

    } else {
        d_uTorsion[threadId] = 0.;
    }


    if (blockSize >= 1024) {
        if (threadId < 512)
            d_uTorsion[threadId] += d_uTorsion[threadId + 512];
        __syncthreads();
    }
    if (blockSize >= 512) {
        if (threadId < 256)
            d_uTorsion[threadId] += d_uTorsion[threadId + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (threadId < 128)
            d_uTorsion[threadId] += d_uTorsion[threadId + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (threadId < 64)
            d_uTorsion[threadId] += d_uTorsion[threadId + 64];
        __syncthreads();
    }

    if (threadId < 32) {
        if (blockSize >= 64) d_uTorsion[threadId] += d_uTorsion[threadId + 32];
        if (blockSize >= 32) d_uTorsion[threadId] += d_uTorsion[threadId + 16];
        if (blockSize >= 16) d_uTorsion[threadId] += d_uTorsion[threadId + 8];
        if (blockSize >= 8) d_uTorsion[threadId] += d_uTorsion[threadId + 4];
        if (blockSize >= 4) d_uTorsion[threadId] += d_uTorsion[threadId + 2];
        if (blockSize >= 2) d_uTorsion[threadId] += d_uTorsion[threadId + 1];

        if(threadId == 0) {
            *d_uT = d_uTorsion[0];

		}
    }

}

__global__ void cudaLJcomp(Particle *particles, int *d_neighbour_list, char *d_sequence, double LV, double *d_uLJVector, int i, int n_mol) {

	extern __shared__ double d_uLJComp[];

	int j = threadIdx.x;
	int blockSize = blockDim.x;
	double r2, u_LJ, f_LJ;
	VectorR dr1;

	if((j < n_mol) && (j > i + 1)) {

    		dr1.x = particles[i].v_r.x - particles[j].v_r.x;
    		dr1.y = particles[i].v_r.y - particles[j].v_r.y;
    		dr1.z = particles[i].v_r.z - particles[j].v_r.z;

    		cudaVerifyBoundary2(&dr1, LV);

    		r2 = (dr1.x * dr1.x) + (dr1.y * dr1.y) + (dr1.z * dr1.z);
    		u_LJ = 4. * (pow(r2, -6) - pow(r2, -3));
    		f_LJ = 24. * (2. * pow(r2, -7) - pow(r2, -4));


            
            if( (d_sequence[i] != 'A') || (d_sequence[j] != 'A') ) {
    			u_LJ = 0.5 * u_LJ;
    			f_LJ = 0.5 * f_LJ;
    		}

    		atomicAdd(&particles[i].v_a.x, (dr1.x * f_LJ));
    		atomicAdd(&particles[i].v_a.y, (dr1.y * f_LJ));
    		atomicAdd(&particles[i].v_a.z, (dr1.z * f_LJ));
    		atomicAdd(&particles[j].v_a.x, -(dr1.x * f_LJ));
    		atomicAdd(&particles[j].v_a.y, -(dr1.y * f_LJ));
    		atomicAdd(&particles[j].v_a.z, -(dr1.z * f_LJ));

    		d_uLJComp[j] = u_LJ;
			
		
    } else {

		d_uLJComp[j] = 0.;
	}
	
    __syncthreads(); 


    if (blockSize >= 1024) {
        if (j < 512){
            d_uLJComp[j] += d_uLJComp[j + 512];}
        __syncthreads();
    }
    if (blockSize >= 512) {
        if (j < 256){
            d_uLJComp[j] += d_uLJComp[j + 256];}
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (j < 128){
            d_uLJComp[j] += d_uLJComp[j + 128];}
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (j < 64){
            d_uLJComp[j] += d_uLJComp[j + 64];}
        __syncthreads();
    }
	
    
        
	if (blockSize >= 64){
		if(j < 32){
			d_uLJComp[j] += d_uLJComp[j + 32];}
	}
	__syncthreads();
	if (blockSize >= 32){
		if(j < 16){
			d_uLJComp[j] += d_uLJComp[j + 16];}
	}
	__syncthreads();
	if (blockSize >= 16){
		if(j < 8){
			d_uLJComp[j] += d_uLJComp[j + 8];}
	}
	__syncthreads();
	if (blockSize >= 8){
		if(j < 4){
			d_uLJComp[j] += d_uLJComp[j + 4];} 
	}
	__syncthreads();
	if (blockSize >= 4){
		if(j < 2){
			d_uLJComp[j] += d_uLJComp[j + 2];}
	}
	__syncthreads();
	if (blockSize >= 2){
		if(j < 1){
			d_uLJComp[j] += d_uLJComp[j + 1];}
	}
	__syncthreads();
	if (blockSize >= 1){
		d_uLJVector[i] = d_uLJComp[0];
	}

	

}

__global__ void cudaLJ(Particle *particles, int *d_neighbour_list, char *d_sequence, double LV, double *d_uT, double *d_uB, double *d_uLJ, double *d_uLJVector, int step, int n_mol){

	extern __shared__ double d_u[];

	// Other Variables
	unsigned int threadId = threadIdx.x;
	unsigned int blockId = blockIdx.x;; 
	unsigned int blockSize = blockDim.x;

	
	if(blockId == 2) {
		if(threadId < n_mol-2)
		{
			__syncthreads();
			if(threadId == 0){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);
			}
			__syncthreads();
			if(threadId < 20){
				if(threadId > 0){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 40){
				if(threadId > 19){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 60){
				if(threadId > 39){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 80){
				if(threadId > 59){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 100){
				if(threadId > 79){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 120){
				if(threadId > 99){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 140){
				if(threadId > 119){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 160){
				if(threadId > 139){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 180){
				if(threadId > 159){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 200){
				if(threadId > 179){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId < 220){
				if(threadId > 199){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);}
			}
			__syncthreads();
			if(threadId > 119){
					cudaLJcomp <<< 1, blockSize, (blockSize + 1) * sizeof(double) >>> (particles, d_neighbour_list, d_sequence, LV, d_uLJVector, threadId, n_mol);
			}
			__syncthreads();
			d_u[threadId] = d_uLJVector[threadId];
			//printf("d_u[%d] = %.8lf\n",threadId, d_u[threadId] );

		}
		else {
			//printf("else - threadId: %d\n", threadId);
			d_u[threadId] = 0.;
		}
	}

}

__global__ void cudaEnergy(Particle *particles, int *d_neighbour_list, char *d_sequence, double LV, double *d_uT, double *d_uB, double *d_uLJ, double *d_uLJVector, int step) {

	extern __shared__ double d_u[];

	// Other Variables
	unsigned int threadId = threadIdx.x;
	unsigned int blockId = blockIdx.x;
	unsigned int blockSize = blockDim.x;
	VectorR dr1, dr2, dr3, a1, a2;
	double cd, c11, c12, c13, c22, c23, c33, pi, qia, qib, cr1, cr2, t1, t2, t3, t4, t5, t6, f;
    //double c

	// ============================================================================================================================ //
	// Bond Energy
	if(blockId == 0) {
		if(threadId < (d_nmol - 2)) {
			dr1.x = particles[threadId + 1].v_r.x - particles[threadId].v_r.x;
			dr1.y = particles[threadId + 1].v_r.y - particles[threadId].v_r.y;
			dr1.z = particles[threadId + 1].v_r.z - particles[threadId].v_r.z;
			dr2.x = particles[threadId + 2].v_r.x - particles[threadId + 1].v_r.x;
			dr2.y = particles[threadId + 2].v_r.y - particles[threadId + 1].v_r.y;
			dr2.z = particles[threadId + 2].v_r.z - particles[threadId + 1].v_r.z;

			cudaVerifyBoundary2(&dr1, LV);  
			cudaVerifyBoundary2(&dr2, LV);

			c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
			c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
			c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;

			cd = sqrt (c11 * c22);


			d_u[threadId] = c12;


			f = -1.0;
			a1.x = (f / cd) * ((c12 / c11) * dr1.x  - dr2.x);
			a1.y = (f / cd) * ((c12 / c11) * dr1.y  - dr2.y);
			a1.z = (f / cd) * ((c12 / c11) * dr1.z  - dr2.z);
			a2.x = (f / cd) * (dr1.x - (c12 / c22) * dr2.x);
			a2.y = (f / cd) * (dr1.y - (c12 / c22) * dr2.y);
			a2.z = (f / cd) * (dr1.z - (c12 / c22) * dr2.z);

			atomicAdd(&particles[threadId].v_a.x, a1.x);
			atomicAdd(&particles[threadId].v_a.y, a1.y);
			atomicAdd(&particles[threadId].v_a.z, a1.z);
			atomicAdd(&particles[threadId + 1].v_a.x, -(a1.x + a2.x));
			atomicAdd(&particles[threadId + 1].v_a.y, -(a1.y + a2.y));
			atomicAdd(&particles[threadId + 1].v_a.z, -(a1.z + a2.z));
			atomicAdd(&particles[threadId + 2].v_a.x, a2.x);
			atomicAdd(&particles[threadId + 2].v_a.y, a2.y);
			atomicAdd(&particles[threadId + 2].v_a.z, a2.z);

		} else {
			d_u[threadId] = 0.;
		}
	}

	// Torsion Energy
	if(blockId == 1) {
		if(threadId < (d_nmol - 3)) {
			dr1.x = particles[threadId + 1].v_r.x - particles[threadId].v_r.x;
			dr1.y = particles[threadId + 1].v_r.y - particles[threadId].v_r.y;
			dr1.z = particles[threadId + 1].v_r.z - particles[threadId].v_r.z;
			dr2.x = particles[threadId + 2].v_r.x - particles[threadId + 1].v_r.x;
			dr2.y = particles[threadId + 2].v_r.y - particles[threadId + 1].v_r.y;
			dr2.z = particles[threadId + 2].v_r.z - particles[threadId + 1].v_r.z;
			dr3.x = particles[threadId + 3].v_r.x - particles[threadId + 2].v_r.x;
			dr3.y = particles[threadId + 3].v_r.y - particles[threadId + 2].v_r.y;
			dr3.z = particles[threadId + 3].v_r.z - particles[threadId + 2].v_r.z;



			cudaVerifyBoundary2(&dr1, LV);
			cudaVerifyBoundary2(&dr2, LV);
			cudaVerifyBoundary2(&dr3, LV);

			c11 = dr1.x * dr1.x + dr1.y * dr1.y + dr1.z * dr1.z;
			c12 = dr1.x * dr2.x + dr1.y * dr2.y + dr1.z * dr2.z;
			c13 = dr1.x * dr3.x + dr1.y * dr3.y + dr1.z * dr3.z;
			c22 = dr2.x * dr2.x + dr2.y * dr2.y + dr2.z * dr2.z;
			c23 = dr2.x * dr3.x + dr2.y * dr3.y + dr2.z * dr3.z;
			c33 = dr3.x * dr3.x + dr3.y * dr3.y + dr3.z * dr3.z;

			pi = c13 * c22 - c12 * c23;
			qia = c11 * c22 - c12 * c12;
			qib = c22 * c33 - c23 * c23;
			cr1 = c12 / c22;
			cr2 = c23 / c22;

			t1 = pi;
			t2 = c11 * c23 - c12 * c13;
			t3 = - qia;
			t4 = qib;
			t5 = c13 * c23 - c12 * c33;
			t6 = - t1;

			d_u[threadId] = (- 0.5) * c13;


			f = 0.5;

			a1.x = f * c22 * (t1 * dr1.x + t2 * dr2.x + t3 * dr3.x) / (sqrt(qia * qib) * qia);
			a1.y = f * c22 * (t1 * dr1.y + t2 * dr2.y + t3 * dr3.y) / (sqrt(qia * qib) * qia);
			a1.z = f * c22 * (t1 * dr1.z + t2 * dr2.z + t3 * dr3.z) / (sqrt(qia * qib) * qia);
			a2.x = f * c22 * (t4 * dr1.x + t5 * dr2.x + t6 * dr3.x) / (sqrt(qia * qib) * qib);
			a2.y = f * c22 * (t4 * dr1.y + t5 * dr2.y + t6 * dr3.y) / (sqrt(qia * qib) * qib);
			a2.z = f * c22 * (t4 * dr1.z + t5 * dr2.z + t6 * dr3.z) / (sqrt(qia * qib) * qib);


			atomicAdd(&particles[threadId].v_a.x, a1.x);
			atomicAdd(&particles[threadId].v_a.y, a1.y);
			atomicAdd(&particles[threadId].v_a.z, a1.z);
			atomicAdd(&particles[threadId + 1].v_a.x, (- (1. + cr1) * a1.x) + cr2 * a2.x);
			atomicAdd(&particles[threadId + 1].v_a.y, (- (1. + cr1) * a1.y) + cr2 * a2.y);
			atomicAdd(&particles[threadId + 1].v_a.z, (- (1. + cr1) * a1.z) + cr2 * a2.z);
			atomicAdd(&particles[threadId + 2].v_a.x, cr1 * a1.x + (- (1. + cr2)) * a2.x);
			atomicAdd(&particles[threadId + 2].v_a.y, cr1 * a1.y + (- (1. + cr2)) * a2.y);
			atomicAdd(&particles[threadId + 2].v_a.z, cr1 * a1.z + (- (1. + cr2)) * a2.z);
			atomicAdd(&particles[threadId + 3].v_a.x, a2.x);
			atomicAdd(&particles[threadId + 3].v_a.y, a2.y);
			atomicAdd(&particles[threadId + 3].v_a.z, a2.z);

		} else {
			d_u[threadId] = 0.;
		}
    }


	if(blockId == 2) {
		if(threadId > (d_nmol - 1)) {

		}

	}


    // ============================================================================================================================ //

	__syncthreads();
	hipDeviceSynchronize();

    if (blockSize >= 1024) {
        if (threadId < 512)
            d_u[threadId] += d_u[threadId + 512];
        __syncthreads();
    }
    if (blockSize >= 512) {
        if (threadId < 256)
            d_u[threadId] += d_u[threadId + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (threadId < 128)
            d_u[threadId] += d_u[threadId + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (threadId < 64)
            d_u[threadId] += d_u[threadId + 64];
        __syncthreads();
    }

    if (threadId < 32) {
        if (blockSize >= 64) d_u[threadId] += d_u[threadId + 32];
		__syncthreads();
        if (blockSize >= 32) d_u[threadId] += d_u[threadId + 16];
		__syncthreads();
        if (blockSize >= 16) d_u[threadId] += d_u[threadId + 8];
		__syncthreads();
        if (blockSize >= 8) d_u[threadId] += d_u[threadId + 4];
		__syncthreads();
        if (blockSize >= 4) d_u[threadId] += d_u[threadId + 2];
		__syncthreads();
        if (blockSize >= 2) d_u[threadId] += d_u[threadId + 1];

        if(threadId == 0) {
			switch (blockId) {
				case 0:
					*d_uB = d_u[0];
					break;
				case 1:
					*d_uT = d_u[0];
					break;
				case 2:
					*d_uLJ = d_u[0];

					break;
			}
		}
    }
}

__global__ void cudaUpdateVelocities(Particle *particles, double dt) {

	for(int i = threadIdx.x; i < d_nmol; i += blockDim.x) {
		particles[i].v_v.x += 0.5 * dt * particles[i].v_a.x;
		particles[i].v_v.y += 0.5 * dt * particles[i].v_a.y;
		particles[i].v_v.z += 0.5 * dt * particles[i].v_a.z;
	}
}

void step() {
    //int i, j;

    hipMemcpy(d_particles, particles, n_mol * sizeof(Particle), hipMemcpyHostToDevice);

	/***   Update positions of all atoms   ***/
	cudaUpdatePos <<< 1, blockSize >>> (d_particles, dt, LV);
    hipDeviceSynchronize();

	cudaUpdateVelocities <<< 1, blockSize >>> (d_particles, dt);
    hipDeviceSynchronize();

	cudaLJ <<< 3, blockSize, (blockSize + 1) * sizeof(double) >>> (d_particles, d_neighbour_list, d_sequence, LV, d_uT, d_uB, d_uLJ, d_uLJVector, i_step, n_mol);
	hipDeviceSynchronize();
    cudaEnergy <<< 3, blockSize, (blockSize + 1) * sizeof(double) >>> (d_particles, d_neighbour_list, d_sequence, LV, d_uT, d_uB, d_uLJ, d_uLJVector, i_step);
    hipDeviceSynchronize();

    

    hipMemcpy(particles, d_particles, n_mol * sizeof(Particle), hipMemcpyDeviceToHost);


	hipMemcpy(&uBond, d_uB, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&uTorsion, d_uT, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&uLJ, d_uLJ, sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

__global__ void cudaShakeRelaxation(Particle *particles, Constraint *constraint, int n_c, double shake_cons_prec, double LV) {
	VectorR dr, dv;
	double cDev, cDevR, cDevV, g, ga;
	int changed, m1, m2, maxCycle;

	maxCycle = 200;
	cDevR = 0;
	cDevV = 0;



	for(int i = blockIdx.x; i < d_nmol; i += gridDim.x) {
		int nCycleR = 0;
		changed = 1;

		while(nCycleR < maxCycle && changed) {
			nCycleR++;
			changed = 0;
			cDev = 0;

			for(int m = threadIdx.x; m < n_c; m += blockDim.x) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;

				cudaVerifyBoundary2(&dr, LV);

				g = (sqr(dr.x) + sqr(dr.y) + sqr(dr.z) - 1.) / 4.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
					changed = 1;
					atomicAdd(&particles[m1].v_r.x, -(g * dr.x));
					atomicAdd(&particles[m1].v_r.y, -(g * dr.y));
					atomicAdd(&particles[m1].v_r.z, -(g * dr.z));

					atomicAdd(&particles[m2].v_r.x, (g * dr.x));
					atomicAdd(&particles[m2].v_r.y, (g * dr.y));
					atomicAdd(&particles[m2].v_r.z, (g * dr.z));

				}
			}
		}
		cDevR = (cDev > cDevR) ? cDev: cDevR;

		int nCycleV = 0;
		changed = 1;

		while(nCycleV < maxCycle && changed) {
			nCycleV++;
			changed = 0;
			cDev = 0;
			for(int m = threadIdx.x; m < n_c; m += blockDim.x) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;

				cudaVerifyBoundary2(&dr, LV);

				dv.x = particles[m1].v_v.x - particles[m2].v_v.x;
				dv.y = particles[m1].v_v.y - particles[m2].v_v.y;
				dv.z = particles[m1].v_v.z - particles[m2].v_v.z;

				g = ((dr.x * dv.x) + (dr.y * dv.y) + (dr.z * dv.z)) / 2.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
					changed = 1;
                    atomicAdd(&particles[m1].v_v.x, -(g * dr.x));
					atomicAdd(&particles[m1].v_v.y, -(g * dr.y));
					atomicAdd(&particles[m1].v_v.z, -(g * dr.z));
					atomicAdd(&particles[m2].v_v.x, (g * dr.x));
					atomicAdd(&particles[m2].v_v.y, (g * dr.y));
					atomicAdd(&particles[m2].v_v.z, (g * dr.z));
				}
			}
		}
		cDevV = (cDev > cDevV) ? cDev : cDevV;
	}
}

void shakeRelaxation() {
	VectorR dr, dv;
	double cDev, cDevR, cDevV, g, ga;
	int changed, m, m1, m2, maxCycle, i;

	maxCycle = 200;
	cDevR = 0;
	cDevV = 0;



	for(i = 0; i < n_mol; i++) {
		nCycleR = 0;
		changed = 1;

		while(nCycleR < maxCycle && changed) {
			nCycleR++;
			changed = 0;
			cDev = 0;

			for(m = 0; m < n_c; m++) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;

				verifyBoundary2(&dr);

				g = (sqr(dr.x) + sqr(dr.y) + sqr(dr.z) - 1.) / 4.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
					changed = 1;
					particles[m1].v_r.x -= (g * dr.x);
					particles[m1].v_r.y -= (g * dr.y);
					particles[m1].v_r.z -= (g * dr.z);

					particles[m2].v_r.x += (g * dr.x);
					particles[m2].v_r.y += (g * dr.y);
					particles[m2].v_r.z += (g * dr.z);

				}
			}
		}

		cDevR = (cDev > cDevR) ? cDev: cDevR;

		nCycleV = 0;
		changed = 1;

		while(nCycleV < maxCycle && changed) {
			nCycleV++;
			changed = 0;
			cDev = 0;
			for(m = 0; m < n_c; m++) {
				m1 = constraint[m].ik;
				m2 = constraint[m].jk;

				dr.x = particles[m1].v_r.x - particles[m2].v_r.x;
				dr.y = particles[m1].v_r.y - particles[m2].v_r.y;
				dr.z = particles[m1].v_r.z - particles[m2].v_r.z;

				verifyBoundary2(&dr);

				dv.x = particles[m1].v_v.x - particles[m2].v_v.x;
				dv.y = particles[m1].v_v.y - particles[m2].v_v.y;
				dv.z = particles[m1].v_v.z - particles[m2].v_v.z;

				g = ((dr.x * dv.x) + (dr.y * dv.y) + (dr.z * dv.z)) / 2.;
				ga = fabs(g);
				cDev = (cDev > ga) ? cDev: ga;

				if(ga > shake_cons_prec) {
					changed = 1;
					particles[m1].v_v.x -= (g * dr.x);
					particles[m1].v_v.y -= (g * dr.y);
					particles[m1].v_v.z -= (g * dr.z);
					particles[m2].v_v.x += (g * dr.x);
					particles[m2].v_v.y += (g * dr.y);
					particles[m2].v_v.z += (g * dr.z);
				}
			}
		}

		cDevV = (cDev > cDevV) ? cDev : cDevV;
	}
}

void berendsenThermostat() {
	double sum, lambda, temp;
	int i;

	sum = 0;
	for(i = 0; i < n_mol; i++) {
		sum += sqr(particles[i].v_v.x) + sqr(particles[i].v_v.y) + sqr(particles[i].v_v.z);
	}

	temp = sum / (3. * (n_mol - 1));
	lambda = sqrt(1. + (dt / c_T) * (temperature / temp - 1.));

	for(i = 0; i < n_mol; i++) {
		particles[i].v_v.x *= lambda;
		particles[i].v_v.y *= lambda;
		particles[i].v_v.z *= lambda;
	}
}

void calcStatus() {
	int i;
	double sum;

	sum = 0.;
	for(i = 0; i < n_mol; i++) {
		sum += sqr(particles[i].v_v.x) + sqr(particles[i].v_v.y) + sqr(particles[i].v_v.z);
	}

	kinetic_energy = 0.5 * sum;
	current_temperature = sum / (3. * (n_mol - 1));
	density = n_mol / (LV * sqr(LV));

	uSum = uTorsion + uBond + uLJ;
	total_energy = uSum + kinetic_energy;

	sum = 0.;
	for(i = 0; i < n_mol - 1; i++) {
		sum += sqrt(sqr(particles[i].v_r.x - particles[i + 1].v_r.x) + sqr(particles[i].v_r.y - particles[i + 1].v_r.y) + sqr(particles[i].v_r.z - particles[i + 1].v_r.z));
 	}
 	bond_avg = sum / (n_mol - 1);
}

void calcRG() {
	int i;
	int h = 0, p = 0;
	double x_avg = 0., y_avg = 0., z_avg = 0.;
	double x_avgH = 0., y_avgH = 0., z_avgH = 0.;
	double x_avgP = 0., y_avgP = 0., z_avgP = 0.;

	rGH = 0.;
	rGP = 0.;
	rG = 0.;
	for(i = 0; i < n_mol; i++) {
		x_avg += particles[i].v_r.x;
		y_avg += particles[i].v_r.y;
		z_avg += particles[i].v_r.z;

		if(sequence[i] == 'A') {
			x_avgH += particles[i].v_r.x;
			y_avgH += particles[i].v_r.y;
			z_avgH += particles[i].v_r.z;

			h++;
		} else {
			x_avgP += particles[i].v_r.x;
			y_avgP += particles[i].v_r.y;
			z_avgP += particles[i].v_r.z;

			p++;
		}
	}

	x_avg = x_avg / n_mol;
	y_avg = y_avg / n_mol;
	z_avg = z_avg / n_mol;

	x_avgH = x_avgH / h;
	y_avgH = y_avgH / h;
	z_avgH = z_avgH / h;

	x_avgP = x_avgP / p;
	y_avgP = y_avgP / p;
	z_avgP = z_avgP / p;

	for(i = 0; i < n_mol; i++) {
		rG += sqr(particles[i].v_r.x - x_avg) + sqr(particles[i].v_r.y - y_avg) + sqr(particles[i].v_r.z - z_avg);

		if(sequence[i] == 'A')
			rGH += sqr(particles[i].v_r.x - x_avgH) + sqr(particles[i].v_r.y - y_avgH) + sqr(particles[i].v_r.z - z_avgH);
		else
			rGP += sqr(particles[i].v_r.x - x_avgP) + sqr(particles[i].v_r.y - y_avgP) + sqr(particles[i].v_r.z - z_avgP);
	}

	rG = sqrt(rG / n_mol);
	rGH = sqrt(rGH / h);
	rGP = sqrt(rGP / p);
}

void evaluate() {
	int i;

	calcStatus();

	if(bond_avg <= 1.) {
		if(i_step == 0) {
			best_potencial_energy = uSum;
			best_step = i_step;
		} else {
			if(uSum < best_potencial_energy) {
				best_potencial_energy = uSum;
				best_step = i_step;

				for(i = 0; i < n_mol; i++) {
					best_structure[i].v_r.x = particles[i].v_r.x;
					best_structure[i].v_r.y = particles[i].v_r.y;
					best_structure[i].v_r.z = particles[i].v_r.z;
				}
			}
		}
	}

	calcRG();
}

void calcCenterMass() {
	int i;
	VectorR r;

	r.x = 0.;
	r.y = 0.;
	r.z = 0.;
	for(i = 0; i < n_mol; i++) {
		r.x += particles[i].v_r.x;
		r.y += particles[i].v_r.y;
		r.z += particles[i].v_r.z;
	}

	center_mass.x = r.x / n_mol;
	center_mass.y = r.y / n_mol;
	center_mass.z = r.z / n_mol;
}

void printSummary(char **argv) {
	FILE *fo;
    char file_name[200];

    //hipMemcpy(particles, d_particles, n_mol * sizeof(Particle), hipMemcpyDeviceToHost);

	calcCenterMass();
	system("clear");
	printf("Step: %d\n\nTemp = %lf\nTotal Lennard-Jones Potential = %lf\nTotal Torsion Potential = %lf\nTotal Chain Angle Potential = %lf\nTotal Potential Energy = %lf\n", i_step, current_temperature, uLJ, uTorsion, uBond, uSum);
	printf("Center of Mass = (%.2lf, %.2lf, %.2lf)\n", center_mass.x, center_mass.y, center_mass.z);
	printf("Radius of Gyration - Hydrophobic = %lf\nRadius of Gyration - Polar = %lf\nRadius of Gyration - All = %lf\n", rGH, rGP, rG);
	printf("Bond Length Average = %lf\n", bond_avg);

	if(print_summary2file == 'y' || print_summary2file == 'Y') {
		sprintf(file_name, "%s_summary_%s.txt", argv[2], argv[3]);
        fo = fopen(file_name, "a+");
		if(i_step == 0)
			fprintf(fo, "Step\tTemperature\tU_LJ\tU_Torsion\tU_ChainAngles\tU_Total\trGH\trGP\trG\n");

		fprintf(fo, "%d\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n", i_step, temperature, uLJ, uTorsion, uBond, uSum, rGH, rGP, rG);
		fclose(fo);
	}
}


void savePathways(char **argv) {
	FILE *fo;
	int i;
    char file_name[200];


    sprintf(file_name, "%s_%s.txt", argv[2], argv[3]);
	fo = fopen(file_name, "a+");

   	fprintf(fo, "N\tx\ty\tz\n");
	for (i = 0; i < n_mol; i++) {
		fprintf(fo, "%d\t%lf\t%lf\t%lf\n", i, particles[i].v_r.x, particles[i].v_r.y, particles[i].v_r.z);
	}

	fprintf(fo, "\n\nPotential Energy = %lf\nStep = %d\n", uSum, i_step);
	fprintf(fo, "uLJ = %lf\nTorsion = %lf\nBond = %lf\n",  uLJ, uTorsion, uBond); 
	fprintf(fo, "rGAll = %lf\nrGH = %lf\nrGP = %lf\n\n\n", rG, rGH, rGP);

	fclose(fo);

}

void freeArrays() {
	free(particles);
	free(best_structure);
	free(constraint);
	free((void *)mMat[0]);
	free((void *)mMat);
}

void freeDevice() {
    hipFree(d_particles);
    hipFree(d_sequence);
    hipFree(d_neighbour_list);
	hipFree(d_uB);
	hipFree(d_uT);
	hipFree(d_uLJ);
    hipFree(d_uLJVector);
}



void finishSim(char **argv) {
	finTimer();
	savePathways(argv);
	freeArrays();
    freeDevice();
}
